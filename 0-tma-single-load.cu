#include "hip/hip_runtime.h"
// TL+ {"platform": "h100"}
// TL+ {"header_files": ["tma-interface.cuh"]}
// TL+ {"compile_flags": ["-lcuda"]}
// TL {"workspace_files": []}

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <random>
#include <stdio.h>

#include "tma-interface.cuh"

// Type alias for bfloat16
typedef __hip_bfloat16 bf16;

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Part 0: Single Block, Single Tile TMA Load
////////////////////////////////////////////////////////////////////////////////

template <int TILE_M, int TILE_N>
__global__ void single_tma_load(__grid_constant__ const CUtensorMap src_map,
                                bf16 *dest) {
    /* TODO: your TMA load code here... */
}

template <int TILE_M, int TILE_N>
void launch_single_tma_load(bf16 *src, bf16 *dest) {
    /* TODO: your launch code here... */
}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

int main() {
    const int M = 64;
    const int N = 128;
    const uint64_t total_size = M * N;

    // Allocate host and device memory
    bf16 *matrix = (bf16 *)malloc(total_size * sizeof(bf16));
    bf16 *d_matrix;
    bf16 *d_dest;
    hipMalloc(&d_matrix, total_size * sizeof(bf16));
    hipMalloc(&d_dest, total_size * sizeof(bf16));

    // Zero out destination buffer
    for (int i = 0; i < total_size; i++) {
        matrix[i] = 0;
    }
    hipMemcpy(d_dest, matrix, total_size * sizeof(bf16),
               hipMemcpyHostToDevice);

    // Initialize source matrix on host
    std::default_random_engine generator(0);
    std::normal_distribution<float> dist(0, 1);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float val = dist(generator);
            matrix[i * N + j] = __float2bfloat16(val);
        }
    }
    hipMemcpy(d_matrix, matrix, total_size * sizeof(bf16),
               hipMemcpyHostToDevice);

    printf("\n\nRunning TMA load kernel...\n\n");

    // Launch the TMA kernel
    launch_single_tma_load<M, N>(d_matrix, d_dest);

    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());

    // Copy result back to host
    bf16 *final_output = (bf16 *)malloc(total_size * sizeof(bf16));
    hipMemcpy(final_output, d_dest, total_size * sizeof(bf16),
               hipMemcpyDeviceToHost);

    // Verify correctness
    bool correct = true;
    for (int x = 0; x < M * N; x++) {
        int i = x / N;
        int j = x % N;
        float ref = (float)matrix[i * N + j];
        float computed = (float)final_output[i * N + j];
        if (ref != computed) {
            correct = false;
            printf("Mismatch at (%d, %d): expected %f, got %f \n", i, j, ref,
                   computed);
            break;
        }
    }

    printf("%s output!\n\n\n", correct ? "Correct" : "Incorrect");

    // Cleanup resources
    hipFree(d_matrix);
    hipFree(d_dest);
    free(matrix);
    free(final_output);

    return 0;
}