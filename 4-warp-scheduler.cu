#include "hip/hip_runtime.h"
// TL+ {"platform": "h100"}
// TL+ {"header_files": ["tma-interface.cuh"]}
// TL+ {"compile_flags": ["-lcuda"]}

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <stdio.h>

#include "tma-interface.cuh"

typedef __hip_bfloat16 bf16;

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Part 4: Bring Your Own Warp Scheduler
////////////////////////////////////////////////////////////////////////////////

__global__ void
tma_multiwarp_pipeline(__grid_constant__ const CUtensorMap tensor_map,
                       __grid_constant__ const CUtensorMap dest_tensor_map,
                       const int N) {
    /* TODO: your TMA memcpy kernel here... */
}

void launch_multiwarp_pipeline(bf16 *dest, bf16 *src, const int N) {
    /*
     * IMPORTANT REQUIREMENT FOR PART 4:
     *
     * To receive credit for this part, you MUST launch the kernel with maximum
     * shared memory allocated.
     *
     * Use hipFuncSetAttribute() with
     * hipFuncAttributeMaxDynamicSharedMemorySize to configure the maximum
     * available shared memory before launching the kernel, and then **launch**
     * it with the maximum amount.
     */

    /* TODO: your launch code here... */
}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

const int elem_per_block = 16384;
__global__ void simple_vector_copy(bf16 *__restrict__ dest,
                                   const bf16 *__restrict__ src, int N) {
    constexpr int VEC_ELEMS = 8;
    using VecT = uint4;

    int total_vecs = elem_per_block / VEC_ELEMS;
    int start_vec = (blockIdx.x * blockDim.x) * total_vecs;

    const VecT *src_vec = reinterpret_cast<const VecT *>(src);
    VecT *dest_vec = reinterpret_cast<VecT *>(dest);

    for (int i = threadIdx.x; i < blockDim.x * total_vecs; i += blockDim.x) {
        dest_vec[start_vec + i] = src_vec[start_vec + i];
    }
}

#define BENCHMARK_KERNEL(kernel_call, num_iters, size_bytes, label)            \
    do {                                                                       \
        hipEvent_t start, stop;                                               \
        CUDA_CHECK(hipEventCreate(&start));                                   \
        CUDA_CHECK(hipEventCreate(&stop));                                    \
        CUDA_CHECK(hipEventRecord(start));                                    \
        for (int i = 0; i < num_iters; i++) {                                  \
            kernel_call;                                                       \
        }                                                                      \
        CUDA_CHECK(hipEventRecord(stop));                                     \
        CUDA_CHECK(hipEventSynchronize(stop));                                \
        float elapsed_time;                                                    \
        CUDA_CHECK(hipEventElapsedTime(&elapsed_time, start, stop));          \
        float time_per_iter = elapsed_time / num_iters;                        \
        float bandwidth_gb_s = (2.0 * size_bytes * 1e-6 / time_per_iter);      \
        printf("%s - Time: %.4f ms, Bandwidth: %.2f GB/s\n", label,            \
               time_per_iter, bandwidth_gb_s);                                 \
        CUDA_CHECK(hipEventDestroy(start));                                   \
        CUDA_CHECK(hipEventDestroy(stop));                                    \
    } while (0)

int main() {
    const size_t size = 132 * 10 * 32 * 128 * 128;

    // Allocate and initialize host memory
    bf16 *matrix = (bf16 *)malloc(size * sizeof(bf16));
    const int N = 128;
    for (int idx = 0; idx < size; idx++) {
        int i = idx / N;
        int j = idx % N;
        float val = fmodf((i * 123 + j * 37) * 0.001f, 2.0f) - 1.0f;
        matrix[idx] = __float2bfloat16(val);
    }

    // Allocate device memory
    bf16 *d_src, *d_dest;
    CUDA_CHECK(hipMalloc(&d_src, size * sizeof(bf16)));
    CUDA_CHECK(hipMalloc(&d_dest, size * sizeof(bf16)));
    CUDA_CHECK(
        hipMemcpy(d_src, matrix, size * sizeof(bf16), hipMemcpyHostToDevice));

    // Test TMA copy correctness
    printf("Testing TMA copy correctness...\n");
    CUDA_CHECK(hipMemset(d_dest, 0, size * sizeof(bf16)));
    launch_multiwarp_pipeline(d_dest, d_src, size);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    bf16 *tma_result = (bf16 *)malloc(size * sizeof(bf16));
    CUDA_CHECK(hipMemcpy(tma_result, d_dest, size * sizeof(bf16),
                          hipMemcpyDeviceToHost));

    bool tma_correct = true;
    for (int idx = 0; idx < size; idx++) {
        if (tma_result[idx] != matrix[idx]) {
            printf("First mismatch at [%d]: %.4f != %.4f\n", idx,
                   __bfloat162float(tma_result[idx]),
                   __bfloat162float(matrix[idx]));
            tma_correct = false;
            break;
        }
    }
    printf("TMA Copy: %s\n\n", tma_correct ? "PASSED" : "FAILED");
    free(tma_result);

    // Test simple copy correctness
    printf("Testing simple copy correctness...\n");
    CUDA_CHECK(hipMemset(d_dest, 0, size * sizeof(bf16)));
    simple_vector_copy<<<size / (elem_per_block * 32), 32>>>(d_dest, d_src,
                                                             size);
    CUDA_CHECK(hipDeviceSynchronize());

    bf16 *simple_result = (bf16 *)malloc(size * sizeof(bf16));
    CUDA_CHECK(hipMemcpy(simple_result, d_dest, size * sizeof(bf16),
                          hipMemcpyDeviceToHost));

    bool simple_correct = true;
    for (int idx = 0; idx < size; idx++) {
        if (simple_result[idx] != matrix[idx]) {
            printf("First mismatch at [%d]: %.4f != %.4f\n", idx,
                   __bfloat162float(tma_result[idx]),
                   __bfloat162float(matrix[idx]));

            simple_correct = false;
            break;
        }
    }
    printf("Simple Copy: %s\n\n", simple_correct ? "PASSED" : "FAILED");
    free(simple_result);

    // Benchmark both kernels
    const int num_iters = 10;
    const size_t size_bytes = size * sizeof(bf16);

    if (tma_correct) {
        BENCHMARK_KERNEL((launch_multiwarp_pipeline(d_dest, d_src, size)),
                         num_iters, size_bytes, "TMA Copy");
    }

    if (simple_correct) {
        BENCHMARK_KERNEL(
            (simple_vector_copy<<<size / (elem_per_block * 32), 32>>>(
                 d_dest, d_src, size),
             hipDeviceSynchronize()),
            num_iters, size_bytes, "Simple Copy");
    }

    // Cleanup
    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dest));
    free(matrix);
    return 0;
}