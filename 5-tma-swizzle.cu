#include "hip/hip_runtime.h"
// TL+ {"platform": "h100"}
// TL+ {"header_files": ["tma-interface.cuh"]}
// TL+ {"compile_flags": ["-lcuda"]}

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <stdio.h>

#include "tma-interface.cuh"

using datatype = uint8_t;

////////////////////////////////////////////////////////////////////////////////
// Part 5: Reverse-Engineering TMA 64B Swizzle
////////////////////////////////////////////////////////////////////////////////

/// <--- your code here --->

template <int TILE_M, int TILE_N, int OFFSET>
__global__ void tma_swizzle(__grid_constant__ const CUtensorMap src_map,
                            datatype *dest) {
    /*
     * IMPORTANT REQUIREMENT FOR PART 5:
     *
     * To get credit, you need to use smem_buffer to store your TMA data.
     * Do not edit the setup for smem_buffer.
     */
    __shared__ alignas(128)
        datatype smem_buffer_abs[TILE_M * TILE_N + 128 * OFFSET];
    datatype *smem_buffer = &smem_buffer_abs[128 * OFFSET];

    // Cast to a "shared pointer" so that it works with
    // cp_async_bulk_tensor_2d_global_to_shared.
    uint32_t smem_addr = __cvta_generic_to_shared(smem_buffer);

    /* TODO: your launch code here... */
}

template <int TILE_M, int TILE_N, int OFFSET>
void launch_tma_swizzle(datatype *src, datatype *dest) {

    /*
     * IMPORTANT REQUIREMENT FOR PART 5:
     *
     * To get credit for this part, launch the tma_swizzle
     * kernel with the CU_TENSOR_MAP_SWIZZLE_64B setting.
     */

    /* TODO: your launch code here... */
}

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

template <int M, int N, int OFFSET>
void set_up_test(datatype *matrix, datatype *d_matrix, datatype *d_dest) {

    printf("Testing offset %d...\n", OFFSET);

    const uint64_t total_size = M * N;
    datatype *final_output = (datatype *)malloc(total_size * sizeof(datatype));
    // Zero out destination buffer
    for (int i = 0; i < total_size; i++) {
        final_output[i] = 0;
    }
    hipMemcpy(d_dest, final_output, total_size * sizeof(datatype),
               hipMemcpyHostToDevice);
    // Launch kernel
    launch_tma_swizzle<M, N, OFFSET>(d_matrix, d_dest);
    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());

    // Copy result back to host
    hipMemcpy(final_output, d_dest, total_size * sizeof(datatype),
               hipMemcpyDeviceToHost);

    // Verify correctness
    bool correct = true;
    for (int x = 0; x < M * N; x++) {
        int i = x / N;
        int j = x % N;
        float ref = (float)matrix[i * N + j];
        float computed = (float)final_output[i * N + j];
        if (ref != computed) {
            correct = false;
            printf("Mismatch at (%d, %d): expected %f, got %f \n", i, j, ref,
                   computed);
            break;
        }
    }

    printf("%s output!\n\n\n", correct ? "Correct" : "Incorrect");

    free(final_output);
}

template <int M, int N>
void run_test(datatype *matrix, datatype *d_matrix, datatype *d_dest) {
    // Test with different offsets
    set_up_test<M, N, 0>(matrix, d_matrix, d_dest);
    set_up_test<M, N, 1>(matrix, d_matrix, d_dest);
    set_up_test<M, N, 2>(matrix, d_matrix, d_dest);
    set_up_test<M, N, 3>(matrix, d_matrix, d_dest);
}

int main() {
    const int M = 1;
    const int N = 64;
    const uint64_t total_size = M * N;

    // Allocate host and device memory
    datatype *matrix = (datatype *)malloc(total_size * sizeof(datatype));
    datatype *d_matrix;
    datatype *d_dest;
    hipMalloc(&d_matrix, total_size * sizeof(datatype));
    hipMalloc(&d_dest, total_size * sizeof(datatype));

    // Initialize source matrix on host
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            matrix[i * N + j] = i + j;
        }
    }
    hipMemcpy(d_matrix, matrix, total_size * sizeof(datatype),
               hipMemcpyHostToDevice);

    printf("\n\nRunning TMA swizzle tests...\n\n");

    run_test<M, N>(matrix, d_matrix, d_dest);

    // Cleanup resources
    hipFree(d_matrix);
    hipFree(d_dest);
    free(matrix);

    return 0;
}