#include "hip/hip_runtime.h"
// TL+ {"platform": "h100"}
// TL+ {"header_files": ["tma-interface.cuh"]}
// TL+ {"compile_flags": ["-lcuda"]}
// TL {"workspace_files": []}

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <random>
#include <stdio.h>

#include "tma-interface.cuh"

// Type alias for bfloat16
typedef __hip_bfloat16 bf16;

/// <--- your code here --->

////////////////////////////////////////////////////////////////////////////////
// Part 2: Single Block, Single Tile TMA Reduce
////////////////////////////////////////////////////////////////////////////////

// Feel free to change the interface to this function if you
// are using a different tile dimension that 2d.
__device__ static __forceinline__ void
cp_async_reduce_add_bulk_tensor_2d_shared_to_global(
    const CUtensorMap *tensor_map, int c0, int c1, const void *src) {
    /* TODO: your TMA reduce intrinsic here... */
}

template <int TILE_M, int TILE_N>
__global__ void
single_tma_reduce(__grid_constant__ const CUtensorMap src_map,
                  __grid_constant__ const CUtensorMap dest_map) {
    /* TODO: your TMA store code here... */
}

template <int TILE_M, int TILE_N>
void launch_single_tma_reduce(bf16 *src, bf16 *dest) {
    /* TODO: your launch code here... */
}

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

int main() {
    const int M = 64;
    const int N = 128;
    const uint64_t total_size = M * N;

    // Allocate host and device memory
    bf16 *matrix = (bf16 *)malloc(total_size * sizeof(bf16));
    bf16 *d_matrix;
    bf16 *d_dest;
    hipMalloc(&d_matrix, total_size * sizeof(bf16));
    hipMalloc(&d_dest, total_size * sizeof(bf16));

    // Copy in 1s for the reduction.
    for (int i = 0; i < total_size; i++) {
        matrix[i] = 1;
    }
    hipMemcpy(d_dest, matrix, total_size * sizeof(bf16),
               hipMemcpyHostToDevice);

    // Initialize source matrix on host
    std::default_random_engine generator(0);
    std::normal_distribution<float> dist(0, 1);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float val = dist(generator);
            matrix[i * N + j] = __float2bfloat16(val);
        }
    }
    hipMemcpy(d_matrix, matrix, total_size * sizeof(bf16),
               hipMemcpyHostToDevice);

    printf("\n\nRunning TMA reduce kernel...\n\n");

    // Launch the TMA kernel
    launch_single_tma_reduce<M, N>(d_matrix, d_dest);

    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());

    // Copy result back to host
    bf16 *final_output = (bf16 *)malloc(total_size * sizeof(bf16));
    hipMemcpy(final_output, d_dest, total_size * sizeof(bf16),
               hipMemcpyDeviceToHost);

    // Verify correctness
    bool correct = true;
    for (int x = 0; x < M * N; x++) {
        int i = x / N;
        int j = x % N;
        float ref = (float)matrix[i * N + j] + 1.0f;
        float computed = (float)final_output[i * N + j];
        float diff = std::fabs(ref - computed);
        if (diff > 0.1) {
            correct = false;
            printf("Mismatch at (%d, %d): expected %f, got %f \n", i, j, ref,
                   computed);
            break;
        }
    }
    printf("%s output!\n\n\n", correct ? "Correct" : "Incorrect");

    // Cleanup resources
    hipFree(d_matrix);
    hipFree(d_dest);
    free(matrix);
    free(final_output);

    return 0;
}